#include "hip/hip_runtime.h"
#include "pix_fmt_conversion.cuh"
#include "hip/hip_runtime.h"

int get_current_time()
{
    struct timeval tv;
    gettimeofday(&tv, NULL); // get milliseconds

    int milliseconds = (tv.tv_sec * 1000 + tv.tv_usec / 1000) % 1000;

    char str_buffer[30];

    time_t raw_time;
    struct tm *time_info;
    time(&raw_time);
    time_info = localtime(&raw_time);
    strftime(str_buffer, sizeof(str_buffer), "%Y-%m-%d %H:%M:%S", time_info);
    sprintf(str_buffer + strlen(str_buffer), ".%03d", milliseconds);

    // return str_buffer;
    printf("%s\n", str_buffer);
    return 0;
}

void __global__ yuv_2_rgb(
    const int *width, // int height,
    const unsigned char *d_yuv_y, const unsigned char *d_yuv_uv, unsigned char *d_rgb)
{

    const int n = blockDim.x * blockIdx.x + threadIdx.x;

    int i = n / (*width);
    int j = n % (*width);

    int u_index = 2 * (((i / 2) * (*width) / 2) + j / 2);
    int v_index = u_index + 1;

    int r = (d_yuv_y[n] - 16) * 1.164f + (d_yuv_uv[v_index] - 128) * 1.793f;
    r = (r > 255) ? 255 : r;
    r = (r > 0) ? r : 0;
    d_rgb[n * 3] = (unsigned char)r;

    int g = (d_yuv_y[n] - 16) * 1.164f - (d_yuv_uv[u_index] - 128) * 0.213f - (d_yuv_uv[v_index] - 128) * 0.533f;
    g = (g > 255) ? 255 : g;
    g = (g > 0) ? g : 0;
    d_rgb[n * 3 + 1] = (unsigned char)g;

    int b = (d_yuv_y[n] - 16) * 1.164f + (d_yuv_uv[u_index] - 128) * 2.112f;
    b = (b > 255) ? 255 : b;
    b = (b > 0) ? b : 0;
    d_rgb[n * 3 + 2] = (unsigned char)(b);
}

void __global__ rgb_2_yuv(
    const int *width, // int height,
    const unsigned char *d_rgb, unsigned char *d_yuv_y, unsigned char *d_yuv_uv)
{
    // int width = 1280;
    // int height = 720;
    const int n = blockDim.x * blockIdx.x + threadIdx.x;

    d_yuv_y[n] = (unsigned char)(16 + 0.183 * d_rgb[3 * n + 2] + 0.614 * d_rgb[3 * n + 1] + 0.062 * d_rgb[3 * n]);

    // index of width(i) and height(j)
    int i = n / (*width);
    int j = n % (*width);

    int u_index = 2 * (((i / 2) * (*width) / 2) + j / 2);
    int v_index = u_index + 1;

    d_yuv_uv[v_index] = (unsigned char)(128 - 0.101 * d_rgb[3 * n + 2] - 0.339 * d_rgb[3 * n + 1] + 0.439 * d_rgb[3 * n]);
    d_yuv_uv[u_index] = (unsigned char)(128 + 0.439 * d_rgb[3 * n + 2] - 0.399 * d_rgb[3 * n + 1] - 0.04 * d_rgb[3 * n]);
}

void initializeCuda(
    int width, int height,
    unsigned char **d_yuv_y, unsigned char **d_yuv_uv, unsigned char **d_rgb,
    int **d_width)
{
    int base_size = width * height / 2;
    // printf("init 1 malloc ... %p\n", (void **)d_yuv_y);
    hipMalloc((void **)d_yuv_y, base_size * 2 * sizeof(unsigned char));
    hipMalloc((void **)d_yuv_uv, base_size * sizeof(unsigned char));
    hipMalloc((void **)d_rgb, base_size * 6 * sizeof(unsigned char));
    hipMalloc((void **)d_width, sizeof(int));
    // printf("init 2 malloc ... %p\n", (void **)d_yuv_y);

    // hipMemcpy(d_width, &width, 4, hipMemcpyHostToDevice);
}

void finalizeCuda(
    unsigned char *d_yuv_y, unsigned char *d_yuv_uv, unsigned char *d_rgb,
    int *d_width)
{
    hipFree(d_yuv_y);
    hipFree(d_yuv_uv);
    hipFree(d_rgb);
    hipFree(d_width);
}

int convertRGB2YUVbyCUDA(
    int width, int height,
    unsigned char *h_yuv_y, unsigned char *h_yuv_uv, unsigned char *h_rgb,
    unsigned char *d_yuv_y, unsigned char *d_yuv_uv, unsigned char *d_rgb,
    int *d_width)
{

    int base_size = width * height / 2;

    const int block_size = 128; // choose between 128 and 256
    const int grid_size = base_size * 2 / block_size;

    hipMemcpy(d_width, &width, 4, hipMemcpyHostToDevice);
    hipMemcpy(d_rgb, h_rgb, base_size * 6 * sizeof(unsigned char), hipMemcpyHostToDevice);

    rgb_2_yuv<<<grid_size, block_size>>>(d_width, d_rgb, d_yuv_y, d_yuv_uv);

    hipMemcpy(h_yuv_y, d_yuv_y, base_size * 2 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(h_yuv_uv, d_yuv_uv, base_size * sizeof(unsigned char), hipMemcpyDeviceToHost);

    return 0;
}

int convertYUV2RGBbyCUDA(
    int width, int height,
    unsigned char *h_yuv_y, unsigned char *h_yuv_uv, unsigned char *h_rgb,
    unsigned char *d_yuv_y, unsigned char *d_yuv_uv, unsigned char *d_rgb,
    int *d_width)
{

    int base_size = width * height / 2;

    const int block_size = 128; // choose between 128 and 256
    const int grid_size = base_size * 2 / block_size;

    // printf("re malloc ... %p\n", (void **)(&d_yuv_y));

    hipMemcpy(d_width, &width, 4, hipMemcpyHostToDevice);
    hipMemcpy(d_yuv_y, h_yuv_y, base_size * 2 * sizeof(unsigned char),
               hipMemcpyHostToDevice);
    hipMemcpy(d_yuv_uv, h_yuv_uv, base_size * sizeof(unsigned char),
               hipMemcpyHostToDevice);

    // ret = get_str_time();
    yuv_2_rgb<<<grid_size, block_size>>>(d_width, d_yuv_y, d_yuv_uv, d_rgb);
    // ret = get_str_time();

    hipMemcpy(h_rgb, d_rgb, base_size * 6 * sizeof(unsigned char),
               hipMemcpyDeviceToHost);
    // ret = get_str_time();

    return 0;
}
