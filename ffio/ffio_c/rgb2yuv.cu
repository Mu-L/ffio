#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>

void __global__ rgb_2_yuv(
    const int *width,
    const unsigned char *d_rgb, unsigned char *d_yuv_y, unsigned char *d_yuv_uv);

const char *dir_name = "/home/cuichengyu/github/ffio/ffio/";

int get_str_time()
{
    struct timeval tv;
    gettimeofday(&tv, NULL); // get milliseconds

    int milliseconds = (tv.tv_sec * 1000 + tv.tv_usec / 1000) % 1000;

    char str_buffer[30];

    time_t raw_time;
    struct tm *time_info;
    time(&raw_time);
    time_info = localtime(&raw_time);
    strftime(str_buffer, sizeof(str_buffer), "%Y-%m-%d %H:%M:%S", time_info);
    sprintf(str_buffer + strlen(str_buffer), ".%03d", milliseconds);

    // return str_buffer;
    printf("%s\n", str_buffer);
    return 0;
}

void __global__ rgb_2_yuv(
    const int *width, // int height,
    const unsigned char *d_rgb, unsigned char *d_yuv_y, unsigned char *d_yuv_uv)
{
    // int width = 1280;
    int height = 720;
    const int n = blockDim.x * blockIdx.x + threadIdx.x;

    d_yuv_y[n] = (unsigned char)(16 + 0.183 * d_rgb[3 * n + 2] + 0.614 * d_rgb[3 * n + 1] + 0.062 * d_rgb[3 * n]);

    // index of width(i) and height(j)
    int i = n / (*width);
    int j = n % (*width);

    int u_index = 2 * (((i / 2) * (*width) / 2) + j / 2);
    int v_index = u_index + 1;

    d_yuv_uv[v_index] = (unsigned char)(128 - 0.101 * d_rgb[3 * n + 2] - 0.339 * d_rgb[3 * n + 1] + 0.439 * d_rgb[3 * n]);
    d_yuv_uv[u_index] = (unsigned char)(128 + 0.439 * d_rgb[3 * n + 2] - 0.399 * d_rgb[3 * n + 1] - 0.04 * d_rgb[3 * n]);
}

int convertRGB2YUV(
    int width, int height,
    unsigned char *h_rgb, unsigned char *h_yuv_y, unsigned char *h_yuv_uv)
{
    int ret;

    // const int width = 1280;
    // const int height = 720;
    int base_size = width * height / 2;

    const int block_size = 128; // choose between 128 and 256
    const int grid_size = base_size * 2 / block_size;

    unsigned char *d_yuv_y, *d_yuv_uv, *d_rgb;
    int *d_width;
    hipMalloc((void **)&d_yuv_y, base_size * 2 * sizeof(unsigned char));
    hipMalloc((void **)&d_yuv_uv, base_size * sizeof(unsigned char));
    hipMalloc((void **)&d_rgb, base_size * 6 * sizeof(unsigned char));
    hipMalloc((void **)&d_width, sizeof(int));

    hipMemcpy(d_width, &width, 4, hipMemcpyHostToDevice);
    hipMemcpy(d_rgb, h_rgb, base_size * 6 * sizeof(unsigned char), hipMemcpyHostToDevice);

    ret = get_str_time();
    rgb_2_yuv<<<grid_size, block_size>>>(d_width, d_rgb, d_yuv_y, d_yuv_uv);
    ret = get_str_time();

    hipMemcpy(h_yuv_y, d_yuv_y, base_size * 2 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(h_yuv_uv, d_yuv_uv, base_size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    ret = get_str_time();

    free(h_yuv_y);
    free(h_yuv_uv);
    free(h_rgb);

    hipFree(d_yuv_y);
    hipFree(d_yuv_uv);
    hipFree(d_rgb);

    return 0;
}

int main(void)
{
    int ret;

    const int width = 1280;
    const int height = 720;
    int base_size = width * height / 2;
    unsigned char *h_yuv_y = (unsigned char *)malloc(base_size * 2);
    unsigned char *h_yuv_uv = (unsigned char *)malloc(base_size);
    unsigned char *h_rgb = (unsigned char *)malloc(base_size * 6);
    unsigned char *h_orig_yuv_y = (unsigned char *)malloc(base_size * 2);
    unsigned char *h_orig_yuv_uv = (unsigned char *)malloc(base_size);

    FILE *f_uv = fopen("/home/cuichengyu/github/ffio/ffio/nv12_uv", "r");
    fread((char *)h_orig_yuv_uv, base_size, 1, f_uv);
    fclose(f_uv);
    FILE *f_y = fopen("/home/cuichengyu/github/ffio/ffio/nv12_y", "r");
    fread((char *)h_orig_yuv_y, base_size * 2, 1, f_y);
    fclose(f_y);

    FILE *f_rgb = fopen("/home/cuichengyu/github/ffio/ffio/rgb", "r");
    fread((char *)h_rgb, base_size * 6, 1, f_rgb);
    fclose(f_rgb);

    ret = get_str_time();

    const int block_size = 128; // choose between 128 and 256
    const int grid_size = base_size * 2 / block_size;

    unsigned char *d_yuv_y, *d_yuv_uv, *d_rgb;
    int *d_width, d_height;
    hipMalloc((void **)&d_yuv_y, base_size * 2 * sizeof(unsigned char));
    hipMalloc((void **)&d_yuv_uv, base_size * sizeof(unsigned char));
    hipMalloc((void **)&d_rgb, base_size * 6 * sizeof(unsigned char));
    hipMalloc((void **)&d_width, sizeof(int));
    ret = get_str_time();

    hipMemcpy(d_width, &width, 4, hipMemcpyHostToDevice);
    // hipMemcpy(d_yuv_y, h_yuv_y, base_size * 2 * sizeof(unsigned char), hipMemcpyHostToDevice);
    // hipMemcpy(d_yuv_uv, h_yuv_uv, base_size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_rgb, h_rgb, base_size * 6 * sizeof(unsigned char), hipMemcpyHostToDevice);

    printf("compute cuda func\n");
    ret = get_str_time();
    rgb_2_yuv<<<grid_size, block_size>>>(d_width, d_rgb, d_yuv_y, d_yuv_uv);

    ret = get_str_time();
    // sleep(15);

    hipMemcpy(h_yuv_y, d_yuv_y, base_size * 2 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(h_yuv_uv, d_yuv_uv, base_size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    ret = get_str_time();

    // FILE *fw = NULL;
    // fw = fopen("/home/cuichengyu/github/ffio/ffio/yuv_y_res", "w");
    // fwrite(h_yuv_y, base_size * 2 * sizeof(unsigned char), 1, fw);
    // fclose(fw);

    // FILE *fw_uv = NULL;
    // fw_uv = fopen("/home/cuichengyu/github/ffio/ffio/yuv_uv_res", "w");
    // fwrite(h_yuv_uv, base_size * sizeof(unsigned char), 1, fw_uv);
    // fclose(fw_uv);

    // int i = 4;
    // int j = 243;
    // int index = i * width + j;
    // printf("index: %d\n", index);
    // printf("width: %d, height: %d.    rgb2yuv: %d, orig_yuv: %d.\n",
    //        i, j, h_yuv_y[index], h_orig_yuv_y[index]);
    // printf("width: %d, height: %d.    rgb2yuv: %d, orig_yuv: %d.\n",
    //        i, j, h_yuv_uv[index], h_orig_yuv_uv[index]);
    // printf("width: %d, height: %d.    rgb2yuv: %d, orig_yuv: %d.\n",
    //        i, j, h_yuv_uv[index + 1], h_orig_yuv_uv[index + 1]);

    free(h_yuv_y);
    free(h_yuv_uv);
    free(h_rgb);
    hipFree(d_yuv_y);
    hipFree(d_yuv_uv);
    hipFree(d_rgb);
    ret = get_str_time();

    return 0;
}
