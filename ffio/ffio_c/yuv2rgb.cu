#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <unistd.h>

void __global__ yuv_2_rgb(
    // int width, int height,
    const unsigned char *d_yuv_y, const unsigned char *d_yuv_uv, unsigned char *d_rgb);
void check(const double *z, const int N);

const char *dir_name = "/home/cuichengyu/github/ffio/ffio/";

// BT.709, which is the standard for HDTV.
float kColorConversion709Default[] = {
    1.164,
    1.164,
    1.164,
    0.0,
    -0.213,
    2.112,
    1.793,
    -0.533,
    0.0,
};

static char str_buffer[128];

int get_str_time()
{
    time_t raw_time;
    struct tm *time_info;
    time(&raw_time);
    time_info = localtime(&raw_time);
    strftime(str_buffer, sizeof(str_buffer), "%Y-%m-%d %H:%M:%S", time_info);
    // return str_buffer;
    printf("%s\n", str_buffer);
    return 0;
}

int main(void)
{
    int ret;

    const int width = 1280;
    const int height = 720;
    int base_size = width * height / 2;
    unsigned char *h_yuv_y = (unsigned char *)malloc(base_size * 2);
    unsigned char *h_yuv_uv = (unsigned char *)malloc(base_size);
    unsigned char *h_rgb = (unsigned char *)malloc(base_size * 6);
    unsigned char *h_orig_rgb = (unsigned char *)malloc(base_size * 6);

    FILE *f_uv = fopen("/home/cuichengyu/github/ffio/ffio/nv12_uv", "r");
    fread((char *)h_yuv_uv, base_size, 1, f_uv);
    fclose(f_uv);
    FILE *f_y = fopen("/home/cuichengyu/github/ffio/ffio/nv12_y", "r");
    fread((char *)h_yuv_y, base_size * 2, 1, f_y);
    fclose(f_y);

    FILE *f_rgb = fopen("/home/cuichengyu/github/ffio/ffio/rgb", "r");
    fread((char *)h_orig_rgb, base_size * 6, 1, f_rgb);
    fclose(f_rgb);

    const int block_size = 128; // choose between 128 and 256
    const int grid_size = base_size * 2 / block_size;

    unsigned char *d_yuv_y, *d_yuv_uv, *d_rgb;
    // int *d_width, d_height;
    hipMalloc((void **)&d_yuv_y, base_size * 2 * sizeof(unsigned char));
    hipMalloc((void **)&d_yuv_uv, base_size * sizeof(unsigned char));
    hipMalloc((void **)&d_rgb, base_size * 6 * sizeof(unsigned char));
    // hipMalloc((void **)&d_width, sizeof(int));

    hipMemcpy(d_yuv_y, h_yuv_y, base_size * 2 * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_yuv_uv, h_yuv_uv, base_size * sizeof(unsigned char), hipMemcpyHostToDevice);

    ret = get_str_time();
    yuv_2_rgb<<<grid_size, block_size>>>(d_yuv_y, d_yuv_uv, d_rgb);

    ret = get_str_time();
    // sleep(15);

    hipMemcpy(h_rgb, d_rgb, base_size * 6 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    ret = get_str_time();

    FILE *fw = NULL;
    fw = fopen("/home/cuichengyu/github/ffio/ffio/rgb_res", "w");
    fwrite(h_rgb, base_size * 6 * sizeof(unsigned char), 1, fw);
    fclose(fw);

    int i = 0;
    int j = 1;
    int index = i * width + j;
    printf("index: %d\n", index);
    printf("width: %d, height: %d.    yuv2rgb: %d, orig_rgb: %d.\n",
           i, j, h_rgb[3 * index], h_orig_rgb[3 * index]);
    printf("width: %d, height: %d.    yuv2rgb: %d, orig_rgb: %d.\n",
           i, j, h_rgb[3 * index + 1], h_orig_rgb[3 * index + 1]);
    printf("width: %d, height: %d.    yuv2rgb: %d, orig_rgb: %d.\n",
           i, j, h_rgb[3 * index + 2], h_orig_rgb[3 * index + 2]);

    free(h_yuv_y);
    free(h_yuv_uv);
    free(h_rgb);
    hipFree(d_yuv_y);
    hipFree(d_yuv_uv);
    hipFree(d_rgb);

    return 0;
}

void __global__ yuv_2_rgb(
    // int width, int height,
    const unsigned char *d_yuv_y, const unsigned char *d_yuv_uv, unsigned char *d_rgb)
{
    int width = 1280;
    int height = 720;
    const int n = blockDim.x * blockIdx.x + threadIdx.x;

    int i = n / width;
    int j = n % width;

    // d_yuv_y[n] -= 16;
    // d_yuv_uv[_n] -= 128;

    int u_index = 2 * (((i / 2) * width / 2) + j / 2);
    int v_index = u_index + 1;

    d_rgb[n * 3 + 2] = (unsigned char)((d_yuv_y[n] - 16) * 1.164f + (d_yuv_uv[v_index] - 128) * 1.793f);
    d_rgb[n * 3 + 1] = (unsigned char)((d_yuv_y[n] - 16) * 1.164f - (d_yuv_uv[u_index] - 128) * 0.213f - (d_yuv_uv[v_index] - 128) * 0.533f);
    d_rgb[n * 3] = (unsigned char)((d_yuv_y[n] - 16) * 1.164f + (d_yuv_uv[u_index] - 128) * 2.112f);

}
